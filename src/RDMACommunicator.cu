#include "RDMACommunicator.hpp"

using namespace std;
using namespace caffe;

RDMACommunicator::RDMACommunicator(int rank, int size) {

  node_rank_ = rank;
  cluster_size_ = size;
    
  adapter_.reset(new RDMAAdapter());
  LOG(INFO)<< "RDMA adapter: " << adapter_->name();

  // The node creates a RDMA address for each node in the cluster except itself.
  // The RDMA addresses are ordered according to the rank of the peers.
  // Create channel for each peer
  peers_.resize(this->cluster_size_);
  for (int i = 0; i < this->cluster_size_; i++) {
    if (i != this->node_rank_)
      peers_[i].reset(new RDMAChannel(*adapter_));
  }

  data_send_.resize(this->cluster_size_);
  data_recv_.resize(this->cluster_size_);

}

RDMACommunicator::~RDMACommunicator() {

  for (int i=0; i<this->cluster_size_; i++){
    peers_[i].reset();
  }
  adapter_.reset();

  CUDA_CHECK(hipFree(data_));
}

void RDMACommunicator::allocateImage(int size){
  size_ = size*peers_.size();
  CUDA_CHECK(hipMalloc(&data_, size));
}

uint8_t* RDMACommunicator::getImage(){
  return (data_ + own_offs_);
}

void RDMACommunicator::localAddresses(vector<string>& vec) {
    vec.resize(this->cluster_size_);
    for (int i = 0; i < this->cluster_size_; i++) {
        if (i != this->node_rank_) {
            vec[i] = peers_[i]->address();
        } else {
            vec[i] = "";
	}
        LOG(INFO) << i << "-th RDMA addr: " << vec[i].c_str();
    }
}

bool RDMACommunicator::connect(vector<const char*>& peer_addresses) {

  //establish RDMA connections
  for (int i = 0; i < this->cluster_size_; i++){
      std::cout << i << std::endl;
        if (i != this->node_rank_) {
	    std::cout << "address: " << peer_addresses[i] << std::endl;
            const char* addr = peer_addresses[i];
            string addr_str(addr, strlen(addr));
            peers_[i]->Connect(addr_str);
        }
  }

  chunk(this->node_rank_, &own_offs_, &own_size_);

  for (int peer = 0; peer < peers_.size(); ++peer) {
    std::cout << "peer: " << peer << ", " << peers_.size() << std::endl;
    if (peer == this->node_rank_) {
      // Chunk for which we are master, connected to all peers. Loops must be
      // imbricated to have buffers created in the same order on all boxes.
      for (int i = 0; i < peers_.size(); ++i) {
        if (i != this->node_rank_) {
          CreateMasterBuffers(i);
        }
      }
    } else {
      // Other chunks are connected to their respective masters
      CreateWorkerBuffers(peer);
    }
  }

  return true;
}

void RDMACommunicator::sync()  {

  if (this->cluster_size_ == 1) return;

  // Send weights to each peer
  int peer = this->node_rank_ + 1;  // To avoid all sending to same peer at the same time
  for (int n = 0; n < peers_.size() - 1; ++n) {
    if (peer == peers_.size()) {
      peer = 0;
    }
    data_send_[peer]->Write();
    peer++;
  }
}

void RDMACommunicator::chunk(int peer, size_t* offs, size_t* size) {
  // TODO align chunks to page size?
  size_t start = (peer + 0) * size_ / peers_.size();
  size_t until = (peer + 1) * size_ / peers_.size();
  *offs = start;
  *size = until - start;
}

void RDMACommunicator::CreateMasterBuffers(int peer) {

  RDMAChannel* channel = peers_[peer].get();
  size_t size = own_size_; //  * sizeof(Dtype);

  std::cout << "CreateMasterBuffers, size: " << size << std::endl;

  // Send data from local (rank_) to remote (peer)
  uint8_t* data = reinterpret_cast<uint8_t*>(data_ + own_offs_);
  data_send_[peer].reset(new RDMABuffer(channel, data, size));

}

void RDMACommunicator::CreateWorkerBuffers(int peer) {

  RDMAChannel* channel = peers_[peer].get();
  size_t offs, size;
  chunk(peer, &offs, &size);
  std::cout << "CreateWorkerBuffers, size: " << size << std::endl;
  // size *= sizeof(Dtype);

  // Recv data from remote (peer) to local (rank_)
  uint8_t* data = reinterpret_cast<uint8_t*>(data_ + offs);
  data_recv_[peer].reset(new RDMABuffer(channel, data, size));

}

void RDMACommunicator::allSum(int & v){
}

void RDMACommunicator::allSum(float * v, int nv){

  float* image = (float*) (data_ + own_offs_);
  for(int i=0; i < nv; i++){
      image[i] = v[i];
  }

  sync();

  for (int n = 0; n < peers_.size() - 1; ++n) {
#ifdef DEBUG
    RDMABuffer* buffer = adapter_->received().pop();
    bool ok = false;
    for (int i = 0; i < data_recv_.size(); ++i) {
      if (buffer == data_recv_[i].get()) {
        ok = true;
      }
    }
    CHECK(ok);
#else
    RDMABuffer* buffer = adapter_->received().pop();
    float* f = (float*) buffer->addr();
    int nf = buffer->size()/sizeof(float);
    std::cout << nv << ", " << nf << ", " << v[nv-1] << ", " << f[nv-1] << std::endl;
    for(int i=0; i < nv; i++){
      v[i] += f[i];
    }
#endif
  }
}

/*
void CommunicatorRDMA::allSum(thrust::device_vector<cusp::complex<float> > & x){
}

void CommunicatorRDMA::allSum(const DeviceRange<cusp::complex<float> > & x){
}

void CommunicatorRDMA::allSum(cusp::array1d<cusp::complex<float>,
     cusp::device_memory> & x){
}

void CommunicatorRDMA::allSum(thrust::device_vector<float > & x){
}

*/
  
